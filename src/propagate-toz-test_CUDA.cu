#include "hip/hip_runtime.h"
/*
nvcc -arch=sm_80 -O3 -DUSE_GPU  -std=c++17 propagate-toz-test_CUDA.cu -L -lcudart   -o ../bin/"propagate_nvcc_cuda" -lnvToolsExt
*/
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <iostream>
#include <chrono>
#include <iomanip>
#include <vector>

#include "nvToolsExt.h"

#define FIXED_RSEED

#ifndef nevts
#define nevts 100
#endif
#ifndef bsize
#define bsize 128
#endif
#ifndef ntrks
#define ntrks 9600 //122880
#endif

#define nb    (ntrks/bsize)
#define smear 0.00001

#ifndef NITER
#define NITER 5 
#endif
#ifndef nlayer
#define nlayer 20
#endif
#ifndef num_streams
#define num_streams 7 //streams changes answers
#endif

#ifndef threadsperblockx
#define threadsperblockx 32
#endif
#define threadsperblocky 512/threadsperblockx
//#define threadsperblocky 1024/threadsperblockx  //unclear why bit 1024 total threads per block gives resource error when running with more than one layer
#ifndef blockspergrid
#define blockspergrid 15
#endif


#define HOSTDEV __host__ __device__

HOSTDEV size_t PosInMtrx(size_t i, size_t j, size_t D) {
  return i*D+j;
}

HOSTDEV size_t SymOffsets33(size_t i) {
  const size_t offs[9] = {0, 1, 3, 1, 2, 4, 3, 4, 5};
  return offs[i];
}

HOSTDEV size_t SymOffsets66(size_t i) {
  const size_t offs[36] = {0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};
  return offs[i];
}


struct ATRK {
  float par[6];
  float cov[21];
  int q;
//  int hitidx[22];
};

struct AHIT {
  float pos[3];
  float cov[6];
};

struct MP1I {
  int data[1*bsize];
};

struct MP22I {
  int data[22*bsize];
};

struct MP3F {
  float data[3*bsize];
};

struct MP6F {
  float data[6*bsize];
};

struct MP3x3 {
  float data[9*bsize];
};
struct MP3x6 {
  float data[18*bsize];
};

struct MP3x3SF {
  float data[6*bsize];
};

struct MP6x6SF {
  float data[21*bsize];
};

struct MP6x6F {
  float data[36*bsize];
};

struct MP2x2SF {
  float data[3*bsize];
};

struct MP2x6 {
  float data[12*bsize];
};

struct MP2F {
  float data[2*bsize];
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;
//  MP22I   hitidx;
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
};



float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0);
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
  call = !call;
  return (mu + sigma * (float) X1);
}

MPTRK* prepareTracks(ATRK inputtrk) {
  MPTRK* result;
  hipMallocManaged((void**)&result,nevts*nb*sizeof(MPTRK)); //fixme, align?
  hipMemAdvise(result,nevts*nb*sizeof(MPTRK),hipMemAdviseSetPreferredLocation,hipCpuDeviceId);
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
        //par
        for (size_t ip=0;ip<6;++ip) {
          result[ib + nb*ie].par.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.par[ip];
        }
        //cov
        for (size_t ip=0;ip<21;++ip) {
	        result[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.cov[ip]*100;
        }
        //q
	      result[ib + nb*ie].q.data[it] = inputtrk.q;//can't really smear this or fit will be wrong
      }
    }
  }
  return result;
}

MPHIT* prepareHits(std::vector<AHIT>& inputhits) {
  //MPHIT* result = (MPHIT*) malloc(nevts*nb*sizeof(MPHIT));
  MPHIT* result;
  hipMallocManaged((void**)&result,nlayer*nevts*nb*sizeof(MPHIT));  //fixme, align?
  hipMemAdvise(result,nlayer*nevts*nb*sizeof(MPHIT),hipMemAdviseSetPreferredLocation,hipCpuDeviceId);
  for (int lay=0;lay<nlayer;++lay) {
    size_t mylay = lay;
    if (lay>=inputhits.size()) {
      // int wraplay = inputhits.size()/lay;
      exit(1);
    }
    AHIT& inputhit = inputhits[mylay];
    for (size_t ie=0;ie<nevts;++ie) {
      for (size_t ib=0;ib<nb;++ib) {
        for (size_t it=0;it<bsize;++it) {
      //pos
          for (size_t ip=0;ip<3;++ip) {
            result[lay+nlayer*(ib + nb*ie)].pos.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.pos[ip];
          }
          //cov
          for (size_t ip=0;ip<6;++ip) {
            result[lay+nlayer*(ib + nb*ie)].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.cov[ip];
          }
        }
      }
    }
  }
  return result;
}


HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib,int layer) {
  return &(tracks[ib + nb*ev+layer*nevts]);
}
HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib,int layer) {
  return &(tracks[ib + nb*ev+layer*nevts]);
}
HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}


HOSTDEV float q(const MP1I* bq, size_t it){
  return (*bq).data[it];
}

HOSTDEV float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it + ipar*bsize];
}
HOSTDEV float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float x    (MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
HOSTDEV float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
HOSTDEV float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
HOSTDEV float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
HOSTDEV float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
HOSTDEV float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
HOSTDEV float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }

HOSTDEV float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}

HOSTDEV float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
HOSTDEV float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
HOSTDEV float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
HOSTDEV float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
HOSTDEV float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
HOSTDEV float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }

HOSTDEV void setpar(MP6F* bpars, size_t it, size_t ipar, float val){
  (*bpars).data[it + ipar*bsize] = val;
}
HOSTDEV void setx    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 0, val); }
HOSTDEV void sety    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 1, val); }
HOSTDEV void setz    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 2, val); }
HOSTDEV void setipt  (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 3, val); }
HOSTDEV void setphi  (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 4, val); }
HOSTDEV void settheta(MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 5, val); }

HOSTDEV void setpar(MPTRK* btracks, size_t it, size_t ipar, float val){
  return setpar(&(*btracks).par,it,ipar,val);
}
HOSTDEV void setx    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 0, val); }
HOSTDEV void sety    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 1, val); }
HOSTDEV void setz    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 2, val); }
HOSTDEV void setipt  (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 3, val); }
HOSTDEV void setphi  (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 4, val); }
HOSTDEV void settheta(MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 5, val); }

HOSTDEV MPHIT* bHit(MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib,int lay) {
  return &(hits[lay + (ib*nlayer) +(ev*nlayer*nb)]);
}

HOSTDEV float pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it + ipar*bsize];
}
HOSTDEV float x(const MP3F* hpos, size_t it)    { return pos(hpos, it, 0); }
HOSTDEV float y(const MP3F* hpos, size_t it)    { return pos(hpos, it, 1); }
HOSTDEV float z(const MP3F* hpos, size_t it)    { return pos(hpos, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t it, size_t ipar){
  return pos(&(*hits).pos,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t it)    { return pos(hits, it, 0); }
HOSTDEV float y(const MPHIT* hits, size_t it)    { return pos(hits, it, 1); }
HOSTDEV float z(const MPHIT* hits, size_t it)    { return pos(hits, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  //[DEBUG by Seyong on Dec. 28, 2020] add 4th argument(nlayer-1) to bHit() below.
  const MPHIT* bhits = bHit(hits, ev, ib, nlayer-1);
  size_t it = tk % bsize;
  return pos(bhits,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 0); }
HOSTDEV float y(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 1); }
HOSTDEV float z(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 2); }



#define N bsize
__forceinline__ __device__ void MultHelixPropEndcap(const MP6x6F* A, const MP6x6SF* B, MP6x6F* C) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = b[ 0*N+n] + a[ 2*N+n]*b[ 3*N+n] + a[ 3*N+n]*b[ 6*N+n] + a[ 4*N+n]*b[10*N+n] + a[ 5*N+n]*b[15*N+n];
    c[ 1*N+n] = b[ 1*N+n] + a[ 2*N+n]*b[ 4*N+n] + a[ 3*N+n]*b[ 7*N+n] + a[ 4*N+n]*b[11*N+n] + a[ 5*N+n]*b[16*N+n];
    c[ 2*N+n] = b[ 3*N+n] + a[ 2*N+n]*b[ 5*N+n] + a[ 3*N+n]*b[ 8*N+n] + a[ 4*N+n]*b[12*N+n] + a[ 5*N+n]*b[17*N+n];
    c[ 3*N+n] = b[ 6*N+n] + a[ 2*N+n]*b[ 8*N+n] + a[ 3*N+n]*b[ 9*N+n] + a[ 4*N+n]*b[13*N+n] + a[ 5*N+n]*b[18*N+n];
    c[ 4*N+n] = b[10*N+n] + a[ 2*N+n]*b[12*N+n] + a[ 3*N+n]*b[13*N+n] + a[ 4*N+n]*b[14*N+n] + a[ 5*N+n]*b[19*N+n];
    c[ 5*N+n] = b[15*N+n] + a[ 2*N+n]*b[17*N+n] + a[ 3*N+n]*b[18*N+n] + a[ 4*N+n]*b[19*N+n] + a[ 5*N+n]*b[20*N+n];
    c[ 6*N+n] = b[ 1*N+n] + a[ 8*N+n]*b[ 3*N+n] + a[ 9*N+n]*b[ 6*N+n] + a[10*N+n]*b[10*N+n] + a[11*N+n]*b[15*N+n];
    c[ 7*N+n] = b[ 2*N+n] + a[ 8*N+n]*b[ 4*N+n] + a[ 9*N+n]*b[ 7*N+n] + a[10*N+n]*b[11*N+n] + a[11*N+n]*b[16*N+n];
    c[ 8*N+n] = b[ 4*N+n] + a[ 8*N+n]*b[ 5*N+n] + a[ 9*N+n]*b[ 8*N+n] + a[10*N+n]*b[12*N+n] + a[11*N+n]*b[17*N+n];
    c[ 9*N+n] = b[ 7*N+n] + a[ 8*N+n]*b[ 8*N+n] + a[ 9*N+n]*b[ 9*N+n] + a[10*N+n]*b[13*N+n] + a[11*N+n]*b[18*N+n];
    c[10*N+n] = b[11*N+n] + a[ 8*N+n]*b[12*N+n] + a[ 9*N+n]*b[13*N+n] + a[10*N+n]*b[14*N+n] + a[11*N+n]*b[19*N+n];
    c[11*N+n] = b[16*N+n] + a[ 8*N+n]*b[17*N+n] + a[ 9*N+n]*b[18*N+n] + a[10*N+n]*b[19*N+n] + a[11*N+n]*b[20*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = 0;
    c[14*N+n] = 0;
    c[15*N+n] = 0;
    c[16*N+n] = 0;
    c[17*N+n] = 0;
    c[18*N+n] = b[ 6*N+n];
    c[19*N+n] = b[ 7*N+n];
    c[20*N+n] = b[ 8*N+n];
    c[21*N+n] = b[ 9*N+n];
    c[22*N+n] = b[13*N+n];
    c[23*N+n] = b[18*N+n];
    c[24*N+n] = a[26*N+n]*b[ 3*N+n] + a[27*N+n]*b[ 6*N+n] + b[10*N+n] + a[29*N+n]*b[15*N+n];
    c[25*N+n] = a[26*N+n]*b[ 4*N+n] + a[27*N+n]*b[ 7*N+n] + b[11*N+n] + a[29*N+n]*b[16*N+n];
    c[26*N+n] = a[26*N+n]*b[ 5*N+n] + a[27*N+n]*b[ 8*N+n] + b[12*N+n] + a[29*N+n]*b[17*N+n];
    c[27*N+n] = a[26*N+n]*b[ 8*N+n] + a[27*N+n]*b[ 9*N+n] + b[13*N+n] + a[29*N+n]*b[18*N+n];
    c[28*N+n] = a[26*N+n]*b[12*N+n] + a[27*N+n]*b[13*N+n] + b[14*N+n] + a[29*N+n]*b[19*N+n];
    c[29*N+n] = a[26*N+n]*b[17*N+n] + a[27*N+n]*b[18*N+n] + b[19*N+n] + a[29*N+n]*b[20*N+n];
    c[30*N+n] = b[15*N+n];
    c[31*N+n] = b[16*N+n];
    c[32*N+n] = b[17*N+n];
    c[33*N+n] = b[18*N+n];
    c[34*N+n] = b[19*N+n];
    c[35*N+n] = b[20*N+n];
  }
}

__forceinline__ __device__ void MultHelixPropTranspEndcap(MP6x6F* A, MP6x6F* B, MP6x6SF* C) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = b[ 0*N+n] + b[ 2*N+n]*a[ 2*N+n] + b[ 3*N+n]*a[ 3*N+n] + b[ 4*N+n]*a[ 4*N+n] + b[ 5*N+n]*a[ 5*N+n];
    c[ 1*N+n] = b[ 6*N+n] + b[ 8*N+n]*a[ 2*N+n] + b[ 9*N+n]*a[ 3*N+n] + b[10*N+n]*a[ 4*N+n] + b[11*N+n]*a[ 5*N+n];
    c[ 2*N+n] = b[ 7*N+n] + b[ 8*N+n]*a[ 8*N+n] + b[ 9*N+n]*a[ 9*N+n] + b[10*N+n]*a[10*N+n] + b[11*N+n]*a[11*N+n];
    c[ 3*N+n] = b[12*N+n] + b[14*N+n]*a[ 2*N+n] + b[15*N+n]*a[ 3*N+n] + b[16*N+n]*a[ 4*N+n] + b[17*N+n]*a[ 5*N+n];
    c[ 4*N+n] = b[13*N+n] + b[14*N+n]*a[ 8*N+n] + b[15*N+n]*a[ 9*N+n] + b[16*N+n]*a[10*N+n] + b[17*N+n]*a[11*N+n];
    c[ 5*N+n] = 0;
    c[ 6*N+n] = b[18*N+n] + b[20*N+n]*a[ 2*N+n] + b[21*N+n]*a[ 3*N+n] + b[22*N+n]*a[ 4*N+n] + b[23*N+n]*a[ 5*N+n];
    c[ 7*N+n] = b[19*N+n] + b[20*N+n]*a[ 8*N+n] + b[21*N+n]*a[ 9*N+n] + b[22*N+n]*a[10*N+n] + b[23*N+n]*a[11*N+n];
    c[ 8*N+n] = 0;
    c[ 9*N+n] = b[21*N+n];
    c[10*N+n] = b[24*N+n] + b[26*N+n]*a[ 2*N+n] + b[27*N+n]*a[ 3*N+n] + b[28*N+n]*a[ 4*N+n] + b[29*N+n]*a[ 5*N+n];
    c[11*N+n] = b[25*N+n] + b[26*N+n]*a[ 8*N+n] + b[27*N+n]*a[ 9*N+n] + b[28*N+n]*a[10*N+n] + b[29*N+n]*a[11*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = b[27*N+n];
    c[14*N+n] = b[26*N+n]*a[26*N+n] + b[27*N+n]*a[27*N+n] + b[28*N+n] + b[29*N+n]*a[29*N+n];
    c[15*N+n] = b[30*N+n] + b[32*N+n]*a[ 2*N+n] + b[33*N+n]*a[ 3*N+n] + b[34*N+n]*a[ 4*N+n] + b[35*N+n]*a[ 5*N+n];
    c[16*N+n] = b[31*N+n] + b[32*N+n]*a[ 8*N+n] + b[33*N+n]*a[ 9*N+n] + b[34*N+n]*a[10*N+n] + b[35*N+n]*a[11*N+n];
    c[17*N+n] = 0;
    c[18*N+n] = b[33*N+n];
    c[19*N+n] = b[32*N+n]*a[26*N+n] + b[33*N+n]*a[27*N+n] + b[34*N+n] + b[35*N+n]*a[29*N+n];
    c[20*N+n] = b[35*N+n];
  }
}

__forceinline__ __device__ void KalmanGainInv(const MP6x6SF* A, const MP3x3SF* B, MP3x3* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the inverse of HpHt +R
  const float* a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const float* b = (*B).data; //ASSUME_ALIGNED(b, 64);
  float* c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    double det =
      ((a[0*N+n]+b[0*N+n])*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])))) -
      ((a[1*N+n]+b[1*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])))) +
      ((a[2*N+n]+b[2*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n]))));
    double invdet = 1.0/det;

    c[ 0*N+n] =  invdet*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 1*N+n] =  -1*invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 2*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 3*N+n] =  -1*invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 4*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 5*N+n] =  -1*invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 6*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n])));
    c[ 7*N+n] =  -1*invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 8*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[6*N+n]+b[3*N+n])) - ((a[1*N+n]+b[1*N+n]) *(a[1*N+n]+b[1*N+n])));
  }
 // __syncthreads();
}

__forceinline__ __device__ void KalmanGain(const MP6x6SF* A, const MP3x3* B, MP3x6* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the kalman gain
  const float* a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const float* b = (*B).data; //ASSUME_ALIGNED(b, 64);
  float* c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = a[0*N+n]*b[0*N+n] + a[1*N+n]*b[3*N+n] + a[2*N+n]*b[6*N+n];
    c[ 1*N+n] = a[0*N+n]*b[1*N+n] + a[1*N+n]*b[4*N+n] + a[2*N+n]*b[7*N+n];
    c[ 2*N+n] = a[0*N+n]*b[2*N+n] + a[1*N+n]*b[5*N+n] + a[2*N+n]*b[8*N+n];
    c[ 3*N+n] = a[1*N+n]*b[0*N+n] + a[6*N+n]*b[3*N+n] + a[7*N+n]*b[6*N+n];
    c[ 4*N+n] = a[1*N+n]*b[1*N+n] + a[6*N+n]*b[4*N+n] + a[7*N+n]*b[7*N+n];
    c[ 5*N+n] = a[1*N+n]*b[2*N+n] + a[6*N+n]*b[5*N+n] + a[7*N+n]*b[8*N+n];
    c[ 6*N+n] = a[2*N+n]*b[0*N+n] + a[7*N+n]*b[3*N+n] + a[11*N+n]*b[6*N+n];
    c[ 7*N+n] = a[2*N+n]*b[1*N+n] + a[7*N+n]*b[4*N+n] + a[11*N+n]*b[7*N+n];
    c[ 8*N+n] = a[2*N+n]*b[2*N+n] + a[7*N+n]*b[5*N+n] + a[11*N+n]*b[8*N+n];
    c[ 9*N+n] = a[3*N+n]*b[0*N+n] + a[8*N+n]*b[3*N+n] + a[12*N+n]*b[6*N+n];
    c[ 10*N+n] = a[3*N+n]*b[1*N+n] + a[8*N+n]*b[4*N+n] + a[12*N+n]*b[7*N+n];
    c[ 11*N+n] = a[3*N+n]*b[2*N+n] + a[8*N+n]*b[5*N+n] + a[12*N+n]*b[8*N+n];
    c[ 12*N+n] = a[4*N+n]*b[0*N+n] + a[9*N+n]*b[3*N+n] + a[13*N+n]*b[6*N+n];
    c[ 13*N+n] = a[4*N+n]*b[1*N+n] + a[9*N+n]*b[4*N+n] + a[13*N+n]*b[7*N+n];
    c[ 14*N+n] = a[4*N+n]*b[2*N+n] + a[9*N+n]*b[5*N+n] + a[13*N+n]*b[8*N+n];
    c[ 15*N+n] = a[5*N+n]*b[0*N+n] + a[10*N+n]*b[3*N+n] + a[14*N+n]*b[6*N+n];
    c[ 16*N+n] = a[5*N+n]*b[1*N+n] + a[10*N+n]*b[4*N+n] + a[14*N+n]*b[7*N+n];
    c[ 17*N+n] = a[5*N+n]*b[2*N+n] + a[10*N+n]*b[5*N+n] + a[14*N+n]*b[8*N+n];
  }
  //__syncthreads();
}

__forceinline__ __device__ void KalmanUpdate(MP6x6SF* trkErr, MP6F* inPar, const MP3x3SF* hitErr, const MP3F* msP){//, MP3x3* inverse_temp, MP3x6* kGain, MP6x6SF* newErr){
  MP3x3 inverse_temp;
  MP3x6 kGain;
  MP6x6SF newErr;
  //MP6F newPar;
  KalmanGainInv(trkErr,hitErr,&inverse_temp);
  //__syncthreads(); 
  KalmanGain(trkErr,&inverse_temp,&kGain);
  //__syncthreads(); 
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    float xin = x(inPar,it);
    float yin = y(inPar,it);
    float zin = z(inPar,it);
    float ptin = 1./ipt(inPar,it);
    float phiin = phi(inPar,it);
    float thetain = theta(inPar,it);
    float xout = x(msP,it);
    float yout = y(msP,it);
    float zout = z(msP,it);
    float ydiff = y(msP,it) - y(inPar,it);
    float xnew = xin + (kGain.data[0*bsize+it]*(xout-xin)) +(kGain.data[1*bsize+it]*(yout-yin)); // removed "zout-zin" term since zin is set to zout thus the term is 0 anyway. 
    float ynew = yin + (kGain.data[3*bsize+it]*(xout-xin)) +(kGain.data[4*bsize+it]*(yout-yin));
    float znew = zin + (kGain.data[6*bsize+it]*(xout-xin)) +(kGain.data[7*bsize+it]*(yout-yin));
    float ptnew = ptin + (kGain.data[9*bsize+it]*(xout-xin)) +(kGain.data[10*bsize+it]*(yout-yin)); 
    float phinew = phiin + (kGain.data[12*bsize+it]*(xout-xin)) +(kGain.data[13*bsize+it]*(yout-yin));
    float thetanew = thetain + (kGain.data[15*bsize+it]*(xout-xin)) +(kGain.data[16*bsize+it]*(yout-yin)); 

    newErr.data[0*bsize+it] = trkErr->data[0*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[0*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[2*bsize+it]);
    newErr.data[1*bsize+it] = trkErr->data[1*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[2*bsize+it] = trkErr->data[2*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[3*bsize+it] = trkErr->data[3*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[4*bsize+it] = trkErr->data[4*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[5*bsize+it] = trkErr->data[5*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[6*bsize+it] = trkErr->data[6*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[7*bsize+it] = trkErr->data[7*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[8*bsize+it] = trkErr->data[8*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[9*bsize+it] = trkErr->data[9*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[10*bsize+it] = trkErr->data[10*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[11*bsize+it] = trkErr->data[11*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[12*bsize+it] = trkErr->data[12*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[13*bsize+it] = trkErr->data[13*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[14*bsize+it] = trkErr->data[14*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[15*bsize+it] = trkErr->data[15*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[16*bsize+it] = trkErr->data[16*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[17*bsize+it] = trkErr->data[17*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[18*bsize+it] = trkErr->data[18*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[19*bsize+it] = trkErr->data[19*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[20*bsize+it] = trkErr->data[20*bsize+it] - (kGain.data[15*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[16*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[17*bsize+it]*trkErr->data[14*bsize+it]);

    setx(inPar,it,xnew );
    sety(inPar,it,ynew );
    setz(inPar,it,znew);
    setipt(inPar,it, ptnew);
    setphi(inPar,it, phinew);
    settheta(inPar,it, thetanew);
  }
  //__syncthreads(); 
  (*trkErr) = newErr;
}


void KalmanUpdate_v2(MP6x6SF* trkErr, MP6F* inPar, const MP3x3SF* hitErr, const MP3F* msP){

   // AddIntoUpperLeft2x2(psErr, msErr, resErr);
   MP2x2SF resErr_loc;
#pragma omp simd
   for (size_t it=0;it<bsize;++it)
   {
     resErr_loc.data[0*bsize+it] = trkErr->data[0*bsize+it] + hitErr->data[0*bsize+it];
     resErr_loc.data[1*bsize+it] = trkErr->data[1*bsize+it] + hitErr->data[1*bsize+it];
     resErr_loc.data[2*bsize+it] = trkErr->data[2*bsize+it] + hitErr->data[2*bsize+it];
   }

   // Matriplex::InvertCramerSym(resErr);
#pragma omp simd
   for (size_t it=0;it<bsize;++it)
   {
     const double det = (double)resErr_loc.data[0*bsize+it] * resErr_loc.data[2*bsize+it] -
                        (double)resErr_loc.data[1*bsize+it] * resErr_loc.data[1*bsize+it];
     const float s   = 1.f / det;
     const float tmp = s * resErr_loc.data[2*bsize+it];
     resErr_loc.data[1*bsize+it] *= -s;
     resErr_loc.data[2*bsize+it]  = s * resErr_loc.data[0*bsize+it];
     resErr_loc.data[0*bsize+it]  = tmp;
   }

   // KalmanGain(psErr, resErr, K);
   MP2x6 kGain;
#pragma omp simd
   for (size_t it=0;it<bsize;++it)
   {
      kGain.data[ 0*bsize+it] = trkErr->data[ 0*bsize+it]*resErr_loc.data[ 0*bsize+it] + trkErr->data[ 1*bsize+it]*resErr_loc.data[ 1*bsize+it];
      kGain.data[ 1*bsize+it] = trkErr->data[ 0*bsize+it]*resErr_loc.data[ 1*bsize+it] + trkErr->data[ 1*bsize+it]*resErr_loc.data[ 2*bsize+it];
      kGain.data[ 2*bsize+it] = trkErr->data[ 1*bsize+it]*resErr_loc.data[ 0*bsize+it] + trkErr->data[ 2*bsize+it]*resErr_loc.data[ 1*bsize+it];
      kGain.data[ 3*bsize+it] = trkErr->data[ 1*bsize+it]*resErr_loc.data[ 1*bsize+it] + trkErr->data[ 2*bsize+it]*resErr_loc.data[ 2*bsize+it];
      kGain.data[ 4*bsize+it] = trkErr->data[ 3*bsize+it]*resErr_loc.data[ 0*bsize+it] + trkErr->data[ 4*bsize+it]*resErr_loc.data[ 1*bsize+it];
      kGain.data[ 5*bsize+it] = trkErr->data[ 3*bsize+it]*resErr_loc.data[ 1*bsize+it] + trkErr->data[ 4*bsize+it]*resErr_loc.data[ 2*bsize+it];
      kGain.data[ 6*bsize+it] = trkErr->data[ 6*bsize+it]*resErr_loc.data[ 0*bsize+it] + trkErr->data[ 7*bsize+it]*resErr_loc.data[ 1*bsize+it];
      kGain.data[ 7*bsize+it] = trkErr->data[ 6*bsize+it]*resErr_loc.data[ 1*bsize+it] + trkErr->data[ 7*bsize+it]*resErr_loc.data[ 2*bsize+it];
      kGain.data[ 8*bsize+it] = trkErr->data[10*bsize+it]*resErr_loc.data[ 0*bsize+it] + trkErr->data[11*bsize+it]*resErr_loc.data[ 1*bsize+it];
      kGain.data[ 9*bsize+it] = trkErr->data[10*bsize+it]*resErr_loc.data[ 1*bsize+it] + trkErr->data[11*bsize+it]*resErr_loc.data[ 2*bsize+it];
      kGain.data[10*bsize+it] = trkErr->data[15*bsize+it]*resErr_loc.data[ 0*bsize+it] + trkErr->data[16*bsize+it]*resErr_loc.data[ 1*bsize+it];
      kGain.data[11*bsize+it] = trkErr->data[15*bsize+it]*resErr_loc.data[ 1*bsize+it] + trkErr->data[16*bsize+it]*resErr_loc.data[ 2*bsize+it];
   }

   // SubtractFirst2(msPar, psPar, res);
   // MultResidualsAdd(K, psPar, res, outPar);
   MP2F res_loc;
#pragma omp simd
   for (size_t it=0;it<bsize;++it)
   {
     res_loc.data[0*bsize+it] =  x(msP,it) - x(inPar,it);
     res_loc.data[1*bsize+it] =  y(msP,it) - y(inPar,it);

     setx    (inPar, it, x    (inPar, it) + kGain.data[ 0*bsize+it] * res_loc.data[ 0*bsize+it] + kGain.data[ 1*bsize+it] * res_loc.data[ 1*bsize+it]);
     sety    (inPar, it, y    (inPar, it) + kGain.data[ 2*bsize+it] * res_loc.data[ 0*bsize+it] + kGain.data[ 3*bsize+it] * res_loc.data[ 1*bsize+it]);
     setz    (inPar, it, z    (inPar, it) + kGain.data[ 4*bsize+it] * res_loc.data[ 0*bsize+it] + kGain.data[ 5*bsize+it] * res_loc.data[ 1*bsize+it]);
     setipt  (inPar, it, ipt  (inPar, it) + kGain.data[ 6*bsize+it] * res_loc.data[ 0*bsize+it] + kGain.data[ 7*bsize+it] * res_loc.data[ 1*bsize+it]);
     setphi  (inPar, it, phi  (inPar, it) + kGain.data[ 8*bsize+it] * res_loc.data[ 0*bsize+it] + kGain.data[ 9*bsize+it] * res_loc.data[ 1*bsize+it]);
     settheta(inPar, it, theta(inPar, it) + kGain.data[10*bsize+it] * res_loc.data[ 0*bsize+it] + kGain.data[11*bsize+it] * res_loc.data[ 1*bsize+it]);
     //note: if ipt changes sign we should update the charge, or we should get rid of the charge altogether and just use the sign of ipt
   }

   // squashPhiMPlex(outPar,N_proc); // ensure phi is between |pi|
   // missing

   // KHC(K, psErr, outErr);
   // outErr.Subtract(psErr, outErr);
   MP6x6SF newErr;
#pragma omp simd
   for (size_t it=0;it<bsize;++it)
   {
      newErr.data[ 0*bsize+it] = kGain.data[ 0*bsize+it]*trkErr->data[ 0*bsize+it] + kGain.data[ 1*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr.data[ 1*bsize+it] = kGain.data[ 2*bsize+it]*trkErr->data[ 0*bsize+it] + kGain.data[ 3*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr.data[ 2*bsize+it] = kGain.data[ 2*bsize+it]*trkErr->data[ 1*bsize+it] + kGain.data[ 3*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr.data[ 3*bsize+it] = kGain.data[ 4*bsize+it]*trkErr->data[ 0*bsize+it] + kGain.data[ 5*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr.data[ 4*bsize+it] = kGain.data[ 4*bsize+it]*trkErr->data[ 1*bsize+it] + kGain.data[ 5*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr.data[ 5*bsize+it] = kGain.data[ 4*bsize+it]*trkErr->data[ 3*bsize+it] + kGain.data[ 5*bsize+it]*trkErr->data[ 4*bsize+it];
      newErr.data[ 6*bsize+it] = kGain.data[ 6*bsize+it]*trkErr->data[ 0*bsize+it] + kGain.data[ 7*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr.data[ 7*bsize+it] = kGain.data[ 6*bsize+it]*trkErr->data[ 1*bsize+it] + kGain.data[ 7*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr.data[ 8*bsize+it] = kGain.data[ 6*bsize+it]*trkErr->data[ 3*bsize+it] + kGain.data[ 7*bsize+it]*trkErr->data[ 4*bsize+it];
      newErr.data[ 9*bsize+it] = kGain.data[ 6*bsize+it]*trkErr->data[ 6*bsize+it] + kGain.data[ 7*bsize+it]*trkErr->data[ 7*bsize+it];
      newErr.data[10*bsize+it] = kGain.data[ 8*bsize+it]*trkErr->data[ 0*bsize+it] + kGain.data[ 9*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr.data[11*bsize+it] = kGain.data[ 8*bsize+it]*trkErr->data[ 1*bsize+it] + kGain.data[ 9*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr.data[12*bsize+it] = kGain.data[ 8*bsize+it]*trkErr->data[ 3*bsize+it] + kGain.data[ 9*bsize+it]*trkErr->data[ 4*bsize+it];
      newErr.data[13*bsize+it] = kGain.data[ 8*bsize+it]*trkErr->data[ 6*bsize+it] + kGain.data[ 9*bsize+it]*trkErr->data[ 7*bsize+it];
      newErr.data[14*bsize+it] = kGain.data[ 8*bsize+it]*trkErr->data[10*bsize+it] + kGain.data[ 9*bsize+it]*trkErr->data[11*bsize+it];
      newErr.data[15*bsize+it] = kGain.data[10*bsize+it]*trkErr->data[ 0*bsize+it] + kGain.data[11*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr.data[16*bsize+it] = kGain.data[10*bsize+it]*trkErr->data[ 1*bsize+it] + kGain.data[11*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr.data[17*bsize+it] = kGain.data[10*bsize+it]*trkErr->data[ 3*bsize+it] + kGain.data[11*bsize+it]*trkErr->data[ 4*bsize+it];
      newErr.data[18*bsize+it] = kGain.data[10*bsize+it]*trkErr->data[ 6*bsize+it] + kGain.data[11*bsize+it]*trkErr->data[ 7*bsize+it];
      newErr.data[19*bsize+it] = kGain.data[10*bsize+it]*trkErr->data[10*bsize+it] + kGain.data[11*bsize+it]*trkErr->data[11*bsize+it];
      newErr.data[20*bsize+it] = kGain.data[10*bsize+it]*trkErr->data[15*bsize+it] + kGain.data[11*bsize+it]*trkErr->data[16*bsize+it];

      newErr.data[ 0*bsize+it] = trkErr->data[ 0*bsize+it] - newErr.data[ 0*bsize+it];
      newErr.data[ 1*bsize+it] = trkErr->data[ 1*bsize+it] - newErr.data[ 1*bsize+it];
      newErr.data[ 2*bsize+it] = trkErr->data[ 2*bsize+it] - newErr.data[ 2*bsize+it];
      newErr.data[ 3*bsize+it] = trkErr->data[ 3*bsize+it] - newErr.data[ 3*bsize+it];
      newErr.data[ 4*bsize+it] = trkErr->data[ 4*bsize+it] - newErr.data[ 4*bsize+it];
      newErr.data[ 5*bsize+it] = trkErr->data[ 5*bsize+it] - newErr.data[ 5*bsize+it];
      newErr.data[ 6*bsize+it] = trkErr->data[ 6*bsize+it] - newErr.data[ 6*bsize+it];
      newErr.data[ 7*bsize+it] = trkErr->data[ 7*bsize+it] - newErr.data[ 7*bsize+it];
      newErr.data[ 8*bsize+it] = trkErr->data[ 8*bsize+it] - newErr.data[ 8*bsize+it];
      newErr.data[ 9*bsize+it] = trkErr->data[ 9*bsize+it] - newErr.data[ 9*bsize+it];
      newErr.data[10*bsize+it] = trkErr->data[10*bsize+it] - newErr.data[10*bsize+it];
      newErr.data[11*bsize+it] = trkErr->data[11*bsize+it] - newErr.data[11*bsize+it];
      newErr.data[12*bsize+it] = trkErr->data[12*bsize+it] - newErr.data[12*bsize+it];
      newErr.data[13*bsize+it] = trkErr->data[13*bsize+it] - newErr.data[13*bsize+it];
      newErr.data[14*bsize+it] = trkErr->data[14*bsize+it] - newErr.data[14*bsize+it];
      newErr.data[15*bsize+it] = trkErr->data[15*bsize+it] - newErr.data[15*bsize+it];
      newErr.data[16*bsize+it] = trkErr->data[16*bsize+it] - newErr.data[16*bsize+it];
      newErr.data[17*bsize+it] = trkErr->data[17*bsize+it] - newErr.data[17*bsize+it];
      newErr.data[18*bsize+it] = trkErr->data[18*bsize+it] - newErr.data[18*bsize+it];
      newErr.data[19*bsize+it] = trkErr->data[19*bsize+it] - newErr.data[19*bsize+it];
      newErr.data[20*bsize+it] = trkErr->data[20*bsize+it] - newErr.data[20*bsize+it];
   }

   (*trkErr) = newErr;
}

const float kfact= 100/(-0.299792458*3.8112);
__device__ __forceinline__ void propagateToZ(const MP6x6SF* inErr, const MP6F* inPar, const MP1I* inChg,const MP3F* msP, 
			  MP6x6SF* outErr, MP6F* outPar, struct MP6x6F* errorProp, struct MP6x6F* temp,const MP3x3SF* hitErr) {
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    const float zout = z(msP,it);
    const float k = q(inChg,it)*kfact;
    const float deltaZ = zout - z(inPar,it);
    const float pt = 1./ipt(inPar,it);
    const float cosP = cosf(phi(inPar,it));
    const float sinP = sinf(phi(inPar,it));
    const float cosT = cosf(theta(inPar,it));
    const float sinT = sinf(theta(inPar,it));
    const float pxin = cosP*pt;
    const float pyin = sinP*pt;
    const float icosT = 1.0/cosT;
    const float icosTk = icosT/k;
    const float alpha = deltaZ*sinT*ipt(inPar,it)*icosTk;///(cosT*k);
    const float sina = sinf(alpha); // this can be approximated;
    const float cosa = cosf(alpha); // this can be approximated;
    setx(outPar,it, x(inPar,it) + k*(pxin*sina - pyin*(1.-cosa)) );
    sety(outPar,it, y(inPar,it) + k*(pyin*sina + pxin*(1.-cosa)) );
    setz(outPar,it,zout);
    setipt(outPar,it, ipt(inPar,it));
    setphi(outPar,it, phi(inPar,it)+alpha );
    settheta(outPar,it, theta(inPar,it) );
    
    const float sCosPsina = sinf(cosP*sina);
    const float cCosPsina = cosf(cosP*sina);
 
    for (size_t i=0;i<6;++i) errorProp->data[bsize*PosInMtrx(i,i,6) + it] = 1.;
    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k*pt)*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ*(icosT*icosTk);   
//    for (size_t i=0;i<6;++i) errorProp->data[bsize*PosInMtrx(i,i,6) + it] = 1.;
//    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)/cosT;
//    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*ipt(inPar,it))-k*(cosP*sina-sinP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
//    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k/ipt(inPar,it))*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
//    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*cosT);
//    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)/cosT;
//    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*ipt(inPar,it))-k*(sinP*sina+cosP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
//    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k/ipt(inPar,it))*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
//    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*cosT);
//    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT/(cosT*k);
//    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ/(cosT*k);
//    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ/(cosT*cosT*k);
  }
  //__syncthreads(); 
  MultHelixPropEndcap(errorProp, inErr, temp);
  //__syncthreads(); 
  MultHelixPropTranspEndcap(errorProp, temp, outErr);
}



__global__ void GPUsequence(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
  int ie_range;
  if(stream == num_streams){ ie_range = (int)(nevts%num_streams);}
  else{ie_range = (int)(nevts/num_streams);}
  for (size_t ie = blockIdx.x; ie<ie_range; ie+=gridDim.x){
    for(size_t ib = threadIdx.y; ib <nb; ib+=blockDim.y){
      const MPTRK* btracks = bTk(trk,ie,ib);
      MPTRK* obtracks = bTk(outtrk,ie,ib);
      for(int layer=0;layer<nlayer;++layer){	
        const MPHIT* bhits = bHit(hit,ie,ib,layer);
      /*__shared__*/ struct MP6x6F errorProp, temp; // using shared here causes a race hazard. idk why i did it this way, might be to include shared. maybe move to inside p2z function 
        propagateToZ(&(*btracks).cov, &(*btracks).par, &(*btracks).q, &(*bhits).pos, 
                     &(*obtracks).cov, &(*obtracks).par, &errorProp, &temp,&(*bhits).cov);
        KalmanUpdate(&(*obtracks).cov,&(*obtracks).par,&(*bhits).cov,&(*bhits).pos);
      }
    }
  }
}



int main (int argc, char* argv[]) {

  printf("RUNNING CUDA!!\n");
  printf("Streams: %d, blocks: %d, threads(x,y): (%d,%d)\n",num_streams,blockspergrid,threadsperblockx,threadsperblocky);
  #include "input_track.h"

  std::vector<AHIT> inputhits{inputhit25,inputhit24,inputhit23,inputhit22,inputhit21,inputhit20,inputhit19,inputhit18,inputhit17,
                              inputhit16,inputhit15,inputhit14,inputhit13,inputhit12,inputhit11,inputhit10,inputhit09,inputhit08,
                              inputhit07,inputhit06,inputhit05,inputhit04,inputhit03,inputhit02,inputhit01,inputhit00};


  printf("track in pos: x=%f, y=%f, z=%f, r=%f, pt=%f, phi=%f, theta=%f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2],
	 sqrtf(inputtrk.par[0]*inputtrk.par[0] + inputtrk.par[1]*inputtrk.par[1]),
	 1./inputtrk.par[3], inputtrk.par[4], inputtrk.par[5]);
  printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66(PosInMtrx(0,0,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(1,1,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(2,2,6))]);
  for (size_t lay=0; lay<nlayer; lay++){
     printf("hit in layer=%lu, pos: x=%f, y=%f, z=%f, r=%f \n", lay, inputhits[lay].pos[0], inputhits[lay].pos[1], inputhits[lay].pos[2], sqrtf(inputhits[lay].pos[0]*inputhits[lay].pos[0] + inputhits[lay].pos[1]*inputhits[lay].pos[1]));
   }
  printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
  printf("NITER=%d\n", NITER);
 
  long setup_start, setup_stop;
  struct timeval timecheck;

  gettimeofday(&timecheck, NULL);
  setup_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
#ifdef FIXED_RSEED
   //[DEBUG by Seyong on Dec. 28, 2020] add an explicit srand(1) call to generate fixed inputs for better debugging.
  srand(1);
#endif

  nvtxRangePushA("GPU init");
  MPTRK* trk = prepareTracks(inputtrk);
  MPHIT* hit = prepareHits(inputhits);
  MPTRK* outtrk;
  hipMallocManaged((void**)&outtrk,nevts*nb*sizeof(MPTRK));
  dim3 grid(blockspergrid,1,1);
  dim3 block(threadsperblockx,threadsperblocky,1); 
  int device = -1;
  hipGetDevice(&device);
  int stream_chunk = ((int)(nevts/num_streams))*nb;//*sizeof(MPTRK);
  int stream_remainder = ((int)(nevts%num_streams))*nb;//*sizeof(MPTRK);
  int stream_range;
  if (stream_remainder == 0){ stream_range =num_streams;}
  else{stream_range = num_streams+1;}
  hipStream_t streams[stream_range];
  for (int s = 0; s<stream_range;s++){
    hipStreamCreate(&streams[s]);
    //hipStreamCreateWithFlags(&streams[s],hipStreamNonBlocking);
  }
  nvtxRangePop();
 
  gettimeofday(&timecheck, NULL);
  setup_stop = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

  printf("done preparing!\n");

  printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*nb*sizeof(struct MPHIT));
  
  auto wall_start = std::chrono::high_resolution_clock::now();
  
  nvtxRangePushA("GPU update");
  for(int itr=0; itr<NITER; itr++){
  for (int s = 0; s<num_streams;s++){
    hipMemPrefetchAsync(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK), device,streams[s]);
      hipMemPrefetchAsync(hit+(s*stream_chunk*nlayer),nlayer*stream_chunk*sizeof(MPHIT), device,streams[s]);
    hipMemAdvise(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK),hipMemAdviseSetPreferredLocation,device);
      hipMemAdvise(hit+(s*stream_chunk*nlayer),nlayer*stream_chunk*sizeof(MPHIT),hipMemAdviseSetPreferredLocation,device);
    //}
    //hipStreamAttachMemAsync(streams[s],trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK),hipMemAttachHost);
    //hipStreamAttachMemAsync(streams[s],hit+(s*stream_chunk),stream_chunk*sizeof(MPHIT),hipMemAttachHost);
    //hipMemAdvise(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK),hipMemAdviseSetReadMostly,device);
    //hipMemAdvise(hit+(s*stream_chunk),stream_chunk*sizeof(MPHIT),hipMemAdviseSetReadMostly,device);
    //hipMemAdvise(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK),hipMemAdviseSetAccessedBy,device);
    //hipMemAdvise(hit+(s*stream_chunk),stream_chunk*sizeof(MPHIT),hipMemAdviseSetAccessedBy,device);
  }
  if(stream_remainder != 0){
    hipMemPrefetchAsync(trk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK), device,streams[num_streams]);
    hipMemAdvise(trk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK),hipMemAdviseSetPreferredLocation,device);
      hipMemPrefetchAsync(hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT), device,streams[num_streams]);
      hipMemAdvise(hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT),hipMemAdviseSetPreferredLocation,device);
  }
//  hipMemAdvise(trk,nevts*nb*sizeof(MPTRK),hipMemAdviseSetPreferredLocation,device);
//  hipMemAdvise(hit,nevts*nb*sizeof(MPHIT),hipMemAdviseSetPreferredLocation,device);
//  hipMemAdvise(trk,nevts*nb*sizeof(MPTRK),hipMemAdviseSetReadMostly,device);
//  hipMemAdvise(hit,nevts*nb*sizeof(MPHIT),hipMemAdviseSetReadMostly,device);


    for (int s = 0; s<num_streams;s++){
  	  GPUsequence<<<grid,block,0,streams[s]>>>(trk+(s*stream_chunk),hit+(s*stream_chunk*nlayer),outtrk+(s*stream_chunk),s);
    }  
    if(stream_remainder != 0){
  	  GPUsequence<<<grid,block,0,streams[num_streams]>>>(trk+(num_streams*stream_chunk),hit+(num_streams*stream_chunk*nlayer),outtrk+(num_streams*stream_chunk),num_streams);
    }  
	  //hipDeviceSynchronize(); // Normal sync

  
    for (int s = 0; s<num_streams;s++){
      hipMemPrefetchAsync(outtrk+(s*stream_chunk),stream_chunk*sizeof(MPTRK), hipCpuDeviceId,streams[s]);
    }
    if(stream_remainder != 0){
      hipMemPrefetchAsync(outtrk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK), hipCpuDeviceId,streams[num_streams]);
    }
  } //end itr loop
  hipDeviceSynchronize(); // shaves a few seconds
  nvtxRangePop();

  auto wall_stop = std::chrono::high_resolution_clock::now();
 
    for (int s = 0; s<stream_range;s++){
      hipStreamDestroy(streams[s]);
    }

   auto wall_diff = wall_stop - wall_start;
   auto wall_time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
   printf("setup time time=%f (s)\n", (setup_stop-setup_start)*0.001);
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), wall_time, wall_time/(nevts*ntrks*int(NITER)));
   printf("formatted %i %i %i %i %i %f 0 %f %i\n",int(NITER),nevts, ntrks, bsize, nb, wall_time, (setup_stop-setup_start)*0.001, num_streams);

    int nnans = 0, nfail = 0;
   float avgx = 0, avgy = 0, avgz = 0;
   float avgpt = 0, avgphi = 0, avgtheta = 0;
   float avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       float hx_ = inputhits[nlayer-1].pos[0];
       float hy_ = inputhits[nlayer-1].pos[1];
       float hz_ = inputhits[nlayer-1].pos[2];
       float hr_ = sqrtf(hx_*hx_ + hy_*hy_);
       if (std::isfinite(x_)==false ||
	   std::isfinite(y_)==false ||
	   std::isfinite(z_)==false ||
	   std::isfinite(pt_)==false ||
	   std::isfinite(phi_)==false ||
	   std::isfinite(theta_)==false
	   ) {
	 nnans++;
	 continue;
       }
       if (fabs( (x_-hx_)/hx_ )>1. ||
	   fabs( (y_-hy_)/hy_ )>1. ||
	   fabs( (z_-hz_)/hz_ )>1. ||
	   fabs( (pt_-12.)/12.)>1.
	   ) {
	 nfail++;
	 continue;
       }
       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
     }
   }
   avgpt = avgpt/float(nevts*ntrks);
   avgphi = avgphi/float(nevts*ntrks);
   avgtheta = avgtheta/float(nevts*ntrks);
   avgx = avgx/float(nevts*ntrks);
   avgy = avgy/float(nevts*ntrks);
   avgz = avgz/float(nevts*ntrks);
   avgdx = avgdx/float(nevts*ntrks);
   avgdy = avgdy/float(nevts*ntrks);
   avgdz = avgdz/float(nevts*ntrks);

   float stdx = 0, stdy = 0, stdz = 0;
   float stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float hx_ = inputhits[nlayer-1].pos[0];
       float hy_ = inputhits[nlayer-1].pos[1];
       float hz_ = inputhits[nlayer-1].pos[2];
       float hr_ = sqrtf(hx_*hx_ + hy_*hy_);
       if (std::isfinite(x_)==false ||
	   std::isfinite(y_)==false ||
	   std::isfinite(z_)==false
	   ) {
	 continue;
       }
       if (fabs( (x_-hx_)/hx_ )>1. ||
	   fabs( (y_-hy_)/hy_ )>1. ||
	   fabs( (z_-hz_)/hz_ )>1. ||
	   fabs( (pt_-12.)/12.)>1.
	   ) {
	 continue;
       }
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
     }
   }

   stdx = sqrtf(stdx/float(nevts*ntrks));
   stdy = sqrtf(stdy/float(nevts*ntrks));
   stdz = sqrtf(stdz/float(nevts*ntrks));
   stddx = sqrtf(stddx/float(nevts*ntrks));
   stddy = sqrtf(stddy/float(nevts*ntrks));
   stddz = sqrtf(stddz/float(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);
  printf("number of tracks with nans=%i\n", nnans);
   printf("number of tracks failed=%i\n", nfail);
	
   hipFree(trk);
   hipFree(hit);
   hipFree(outtrk);
   
return 0;
}


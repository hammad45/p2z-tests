#include "hip/hip_runtime.h"
/*
icc propagate-toz-test.C -o propagate-toz-test.exe -fopenmp -O3
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <alpaka/alpaka.hpp>
#include <functional>
#include <iostream>
#include <cstdint>
#include <chrono>
#include <iomanip>

#define FIXED_RSEED

#ifndef bsize
#define bsize 128
#endif
#ifndef ntrks
#define ntrks 9600
#endif

#define nb    (ntrks/bsize)
#ifndef nevts
#define nevts 100
#endif
#define smear 0.1

#ifndef NITER
#define NITER 5 
#endif
#ifndef nlayer
#define nlayer 20
#endif

#define HOSTDEV __host__ __device__


HOSTDEV size_t PosInMtrx(size_t i, size_t j, size_t D) {
  return i*D+j;
}

HOSTDEV size_t SymOffsets33(size_t i) {
  const size_t offs[9] = {0, 1, 3, 1, 2, 4, 3, 4, 5};
  return offs[i];
}

HOSTDEV size_t SymOffsets66(size_t i) {
  const size_t offs[36] = {0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};
  return offs[i];
}

struct ATRK {
  float par[6];
  float cov[21];
  int q;
//  int hitidx[22];
};

struct AHIT {
  float pos[3];
  float cov[6];
};

struct MP1I {
  int data[1*bsize];
};

struct MP22I {
  int data[22*bsize];
};

struct MP3F {
  float data[3*bsize];
};

struct MP6F {
  float data[6*bsize];
};

struct MP3x3 {
  float data[9*bsize];
};
struct MP3x6 {
  float data[18*bsize];
};

struct MP3x3SF {
  float data[6*bsize];
};

struct MP6x6SF {
  float data[21*bsize];
};

struct MP6x6F {
  float data[36*bsize];
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;
//  MP22I   hitidx;
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
};

float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0); 
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult; 
  call = !call; 
  return (mu + sigma * (float) X1);
}

HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
   return &(tracks[ib + nb*ev]);
}
 
HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
   return &(tracks[ib + nb*ev]);
}
 
HOSTDEV float q(const MP1I* bq, size_t it){
  return (*bq).data[it];
}

HOSTDEV float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it + ipar*bsize];
}
HOSTDEV float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
HOSTDEV float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
HOSTDEV float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
HOSTDEV float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
HOSTDEV float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
HOSTDEV float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
HOSTDEV float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }

HOSTDEV float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}
HOSTDEV float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
HOSTDEV float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
HOSTDEV float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
HOSTDEV float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
HOSTDEV float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
HOSTDEV float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }

HOSTDEV void setpar(MP6F* bpars, size_t it, size_t ipar, float val){
  (*bpars).data[it + ipar*bsize] = val;
}
HOSTDEV void setx    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 0, val); }
HOSTDEV void sety    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 1, val); }
HOSTDEV void setz    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 2, val); }
HOSTDEV void setipt  (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 3, val); }
HOSTDEV void setphi  (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 4, val); }
HOSTDEV void settheta(MP6F* bpars, size_t it, float val){ setpar(bpars, it, 5, val); }

HOSTDEV void setpar(MPTRK* btracks, size_t it, size_t ipar, float val){
  setpar(&(*btracks).par,it,ipar,val);
}
HOSTDEV void setx    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 0, val); }
HOSTDEV void sety    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 1, val); }
HOSTDEV void setz    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 2, val); }
HOSTDEV void setipt  (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 3, val); }
HOSTDEV void setphi  (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 4, val); }
HOSTDEV void settheta(MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 5, val); }
 
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib,size_t lay) {
  return &(hits[lay + (ib*nlayer) +(ev*nlayer*nb)]);
}

HOSTDEV float pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it + ipar*bsize];
}
HOSTDEV float x(const MP3F* hpos, size_t it)    { return pos(hpos, it, 0); }
HOSTDEV float y(const MP3F* hpos, size_t it)    { return pos(hpos, it, 1); }
HOSTDEV float z(const MP3F* hpos, size_t it)    { return pos(hpos, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t it, size_t ipar){
  return pos(&(*hits).pos,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t it)    { return pos(hits, it, 0); }
HOSTDEV float y(const MPHIT* hits, size_t it)    { return pos(hits, it, 1); }
HOSTDEV float z(const MPHIT* hits, size_t it)    { return pos(hits, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  //[DEBUG by Seyong on Dec. 28, 2020] add 4th argument(nlayer-1) to bHit() below.
  const MPHIT* bhits = bHit(hits, ev, ib, nlayer-1);
  size_t it = tk % bsize;
  return pos(bhits,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 0); }
HOSTDEV float y(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 1); }
HOSTDEV float z(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 2); }

MPTRK* prepareTracks(ATRK inputtrk) {
  //MPTRK* result = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK)); //fixme, align?
  MPTRK* result;
  hipHostMalloc((void**)&result,nevts*nb*sizeof(MPTRK));
  // store in element order for bunches of bsize matrices (a la matriplex)
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
      	//par
      	for (size_t ip=0;ip<6;++ip) {
      	  result[ib + nb*ie].par.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.par[ip];
      	}
      	//cov
      	for (size_t ip=0;ip<21;++ip) {
      	  result[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.cov[ip];
      	}
      	//q
      	result[ib + nb*ie].q.data[it] = inputtrk.q-2*ceil(-0.5 + (float)rand() / RAND_MAX);//fixme check
      }
    }
  }
  return result;
}

MPHIT* prepareHits(AHIT inputhit) {
 // MPHIT* result = (MPHIT*) malloc(nlayer*nevts*nb*sizeof(MPHIT));  //fixme, align?
  MPHIT* result;
  hipHostMalloc((void**)&result,nlayer*nevts*nb*sizeof(MPHIT));
  // store in element order for bunches of bsize matrices (a la matriplex)
  for (size_t lay=0;lay<nlayer;++lay) {
    for (size_t ie=0;ie<nevts;++ie) {
      for (size_t ib=0;ib<nb;++ib) {
        for (size_t it=0;it<bsize;++it) {
        	//pos
        	for (size_t ip=0;ip<3;++ip) {
        	  result[lay+nlayer*(ib + nb*ie)].pos.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.pos[ip];
        	}
        	//cov
        	for (size_t ip=0;ip<6;++ip) {
        	  result[lay+nlayer*(ib + nb*ie)].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.cov[ip];
        	}
        }
      }
    }
  }
  return result;
}

#define N bsize
template< typename TAcc>
__host__ __device__ inline void MultHelixPropEndcap(const MP6x6F* A, const MP6x6SF* B, MP6x6F* C, TAcc const & acc) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
  using Dim = alpaka::dim::Dim<TAcc>;
  using Idx = alpaka::idx::Idx<TAcc>;
  using Vec = alpaka::vec::Vec<Dim, Idx>;

  Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
  Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
  for (int n = threadIdx[0]; n < N; n+=threadExtent[0])
  {
    c[ 0*N+n] = b[ 0*N+n] + a[ 2*N+n]*b[ 3*N+n] + a[ 3*N+n]*b[ 6*N+n] + a[ 4*N+n]*b[10*N+n] + a[ 5*N+n]*b[15*N+n];
    c[ 1*N+n] = b[ 1*N+n] + a[ 2*N+n]*b[ 4*N+n] + a[ 3*N+n]*b[ 7*N+n] + a[ 4*N+n]*b[11*N+n] + a[ 5*N+n]*b[16*N+n];
    c[ 2*N+n] = b[ 3*N+n] + a[ 2*N+n]*b[ 5*N+n] + a[ 3*N+n]*b[ 8*N+n] + a[ 4*N+n]*b[12*N+n] + a[ 5*N+n]*b[17*N+n];
    c[ 3*N+n] = b[ 6*N+n] + a[ 2*N+n]*b[ 8*N+n] + a[ 3*N+n]*b[ 9*N+n] + a[ 4*N+n]*b[13*N+n] + a[ 5*N+n]*b[18*N+n];
    c[ 4*N+n] = b[10*N+n] + a[ 2*N+n]*b[12*N+n] + a[ 3*N+n]*b[13*N+n] + a[ 4*N+n]*b[14*N+n] + a[ 5*N+n]*b[19*N+n];
    c[ 5*N+n] = b[15*N+n] + a[ 2*N+n]*b[17*N+n] + a[ 3*N+n]*b[18*N+n] + a[ 4*N+n]*b[19*N+n] + a[ 5*N+n]*b[20*N+n];
    c[ 6*N+n] = b[ 1*N+n] + a[ 8*N+n]*b[ 3*N+n] + a[ 9*N+n]*b[ 6*N+n] + a[10*N+n]*b[10*N+n] + a[11*N+n]*b[15*N+n];
    c[ 7*N+n] = b[ 2*N+n] + a[ 8*N+n]*b[ 4*N+n] + a[ 9*N+n]*b[ 7*N+n] + a[10*N+n]*b[11*N+n] + a[11*N+n]*b[16*N+n];
    c[ 8*N+n] = b[ 4*N+n] + a[ 8*N+n]*b[ 5*N+n] + a[ 9*N+n]*b[ 8*N+n] + a[10*N+n]*b[12*N+n] + a[11*N+n]*b[17*N+n];
    c[ 9*N+n] = b[ 7*N+n] + a[ 8*N+n]*b[ 8*N+n] + a[ 9*N+n]*b[ 9*N+n] + a[10*N+n]*b[13*N+n] + a[11*N+n]*b[18*N+n];
    c[10*N+n] = b[11*N+n] + a[ 8*N+n]*b[12*N+n] + a[ 9*N+n]*b[13*N+n] + a[10*N+n]*b[14*N+n] + a[11*N+n]*b[19*N+n];
    c[11*N+n] = b[16*N+n] + a[ 8*N+n]*b[17*N+n] + a[ 9*N+n]*b[18*N+n] + a[10*N+n]*b[19*N+n] + a[11*N+n]*b[20*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = 0;
    c[14*N+n] = 0;
    c[15*N+n] = 0;
    c[16*N+n] = 0;
    c[17*N+n] = 0;
    c[18*N+n] = b[ 6*N+n];
    c[19*N+n] = b[ 7*N+n];
    c[20*N+n] = b[ 8*N+n];
    c[21*N+n] = b[ 9*N+n];
    c[22*N+n] = b[13*N+n];
    c[23*N+n] = b[18*N+n];
    c[24*N+n] = a[26*N+n]*b[ 3*N+n] + a[27*N+n]*b[ 6*N+n] + b[10*N+n] + a[29*N+n]*b[15*N+n];
    c[25*N+n] = a[26*N+n]*b[ 4*N+n] + a[27*N+n]*b[ 7*N+n] + b[11*N+n] + a[29*N+n]*b[16*N+n];
    c[26*N+n] = a[26*N+n]*b[ 5*N+n] + a[27*N+n]*b[ 8*N+n] + b[12*N+n] + a[29*N+n]*b[17*N+n];
    c[27*N+n] = a[26*N+n]*b[ 8*N+n] + a[27*N+n]*b[ 9*N+n] + b[13*N+n] + a[29*N+n]*b[18*N+n];
    c[28*N+n] = a[26*N+n]*b[12*N+n] + a[27*N+n]*b[13*N+n] + b[14*N+n] + a[29*N+n]*b[19*N+n];
    c[29*N+n] = a[26*N+n]*b[17*N+n] + a[27*N+n]*b[18*N+n] + b[19*N+n] + a[29*N+n]*b[20*N+n];
    c[30*N+n] = b[15*N+n];
    c[31*N+n] = b[16*N+n];
    c[32*N+n] = b[17*N+n];
    c[33*N+n] = b[18*N+n];
    c[34*N+n] = b[19*N+n];
    c[35*N+n] = b[20*N+n];
  }
}

template< typename TAcc>
__host__ __device__ inline void MultHelixPropTranspEndcap(const MP6x6F* A, const MP6x6F* B, MP6x6SF* C, TAcc const & acc) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
  using Dim = alpaka::dim::Dim<TAcc>;
  using Idx = alpaka::idx::Idx<TAcc>;
  using Vec = alpaka::vec::Vec<Dim, Idx>;

  Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
  Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
  for (int n = threadIdx[0]; n < N; n+=threadExtent[0])
  {
    c[ 0*N+n] = b[ 0*N+n] + b[ 2*N+n]*a[ 2*N+n] + b[ 3*N+n]*a[ 3*N+n] + b[ 4*N+n]*a[ 4*N+n] + b[ 5*N+n]*a[ 5*N+n];
    c[ 1*N+n] = b[ 6*N+n] + b[ 8*N+n]*a[ 2*N+n] + b[ 9*N+n]*a[ 3*N+n] + b[10*N+n]*a[ 4*N+n] + b[11*N+n]*a[ 5*N+n];
    c[ 2*N+n] = b[ 7*N+n] + b[ 8*N+n]*a[ 8*N+n] + b[ 9*N+n]*a[ 9*N+n] + b[10*N+n]*a[10*N+n] + b[11*N+n]*a[11*N+n];
    c[ 3*N+n] = b[12*N+n] + b[14*N+n]*a[ 2*N+n] + b[15*N+n]*a[ 3*N+n] + b[16*N+n]*a[ 4*N+n] + b[17*N+n]*a[ 5*N+n];
    c[ 4*N+n] = b[13*N+n] + b[14*N+n]*a[ 8*N+n] + b[15*N+n]*a[ 9*N+n] + b[16*N+n]*a[10*N+n] + b[17*N+n]*a[11*N+n];
    c[ 5*N+n] = 0;
    c[ 6*N+n] = b[18*N+n] + b[20*N+n]*a[ 2*N+n] + b[21*N+n]*a[ 3*N+n] + b[22*N+n]*a[ 4*N+n] + b[23*N+n]*a[ 5*N+n];
    c[ 7*N+n] = b[19*N+n] + b[20*N+n]*a[ 8*N+n] + b[21*N+n]*a[ 9*N+n] + b[22*N+n]*a[10*N+n] + b[23*N+n]*a[11*N+n];
    c[ 8*N+n] = 0;
    c[ 9*N+n] = b[21*N+n];
    c[10*N+n] = b[24*N+n] + b[26*N+n]*a[ 2*N+n] + b[27*N+n]*a[ 3*N+n] + b[28*N+n]*a[ 4*N+n] + b[29*N+n]*a[ 5*N+n];
    c[11*N+n] = b[25*N+n] + b[26*N+n]*a[ 8*N+n] + b[27*N+n]*a[ 9*N+n] + b[28*N+n]*a[10*N+n] + b[29*N+n]*a[11*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = b[27*N+n];
    c[14*N+n] = b[26*N+n]*a[26*N+n] + b[27*N+n]*a[27*N+n] + b[28*N+n] + b[29*N+n]*a[29*N+n];
    c[15*N+n] = b[30*N+n] + b[32*N+n]*a[ 2*N+n] + b[33*N+n]*a[ 3*N+n] + b[34*N+n]*a[ 4*N+n] + b[35*N+n]*a[ 5*N+n];
    c[16*N+n] = b[31*N+n] + b[32*N+n]*a[ 8*N+n] + b[33*N+n]*a[ 9*N+n] + b[34*N+n]*a[10*N+n] + b[35*N+n]*a[11*N+n];
    c[17*N+n] = 0;
    c[18*N+n] = b[33*N+n];
    c[19*N+n] = b[32*N+n]*a[26*N+n] + b[33*N+n]*a[27*N+n] + b[34*N+n] + b[35*N+n]*a[29*N+n];
    c[20*N+n] = b[35*N+n];
  }
}

template< typename TAcc>
__device__ inline void KalmanGainInv(const MP6x6SF* A, const MP3x3SF* B, MP3x3* C, TAcc const & acc) {
  const float* a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const float* b = (*B).data; //ASSUME_ALIGNED(b, 64);
  float* c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  using Dim = alpaka::dim::Dim<TAcc>;
  using Idx = alpaka::idx::Idx<TAcc>;
  using Vec = alpaka::vec::Vec<Dim, Idx>;

  Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
  Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
  for (int n = threadIdx[0]; n < N; n+=threadExtent[0])
  {
    double det =
      ((a[0*N+n]+b[0*N+n])*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])))) -
      ((a[1*N+n]+b[1*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])))) +
      ((a[2*N+n]+b[2*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n]))));
    double invdet = 1.0/det;

    c[ 0*N+n] =  invdet*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 1*N+n] =  -1*invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 2*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 3*N+n] =  -1*invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 4*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 5*N+n] =  -1*invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 6*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n])));
    c[ 7*N+n] =  -1*invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 8*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[6*N+n]+b[3*N+n])) - ((a[1*N+n]+b[1*N+n]) *(a[1*N+n]+b[1*N+n])));
  }
}

template< typename TAcc>
__device__ inline void KalmanGain(const MP6x6SF* A, const MP3x3* B, MP3x6* C, TAcc const & acc) {
  const float* a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const float* b = (*B).data; //ASSUME_ALIGNED(b, 64);
  float* c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  using Dim = alpaka::dim::Dim<TAcc>;
  using Idx = alpaka::idx::Idx<TAcc>;
  using Vec = alpaka::vec::Vec<Dim, Idx>;

  Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
  Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
  for (int n = threadIdx[0]; n < N; n+=threadExtent[0])
  {
    c[ 0*N+n] = a[0*N+n]*b[0*N+n] + a[1*N+n]*b[3*N+n] + a[2*N+n]*b[6*N+n];
    c[ 1*N+n] = a[0*N+n]*b[1*N+n] + a[1*N+n]*b[4*N+n] + a[2*N+n]*b[7*N+n];
    c[ 2*N+n] = a[0*N+n]*b[2*N+n] + a[1*N+n]*b[5*N+n] + a[2*N+n]*b[8*N+n];
    c[ 3*N+n] = a[1*N+n]*b[0*N+n] + a[6*N+n]*b[3*N+n] + a[7*N+n]*b[6*N+n];
    c[ 4*N+n] = a[1*N+n]*b[1*N+n] + a[6*N+n]*b[4*N+n] + a[7*N+n]*b[7*N+n];
    c[ 5*N+n] = a[1*N+n]*b[2*N+n] + a[6*N+n]*b[5*N+n] + a[7*N+n]*b[8*N+n];
    c[ 6*N+n] = a[2*N+n]*b[0*N+n] + a[7*N+n]*b[3*N+n] + a[11*N+n]*b[6*N+n];
    c[ 7*N+n] = a[2*N+n]*b[1*N+n] + a[7*N+n]*b[4*N+n] + a[11*N+n]*b[7*N+n];
    c[ 8*N+n] = a[2*N+n]*b[2*N+n] + a[7*N+n]*b[5*N+n] + a[11*N+n]*b[8*N+n];
    c[ 9*N+n] = a[3*N+n]*b[0*N+n] + a[8*N+n]*b[3*N+n] + a[12*N+n]*b[6*N+n];
    c[ 10*N+n] = a[3*N+n]*b[1*N+n] + a[8*N+n]*b[4*N+n] + a[12*N+n]*b[7*N+n];
    c[ 11*N+n] = a[3*N+n]*b[2*N+n] + a[8*N+n]*b[5*N+n] + a[12*N+n]*b[8*N+n];
    c[ 12*N+n] = a[4*N+n]*b[0*N+n] + a[9*N+n]*b[3*N+n] + a[13*N+n]*b[6*N+n];
    c[ 13*N+n] = a[4*N+n]*b[1*N+n] + a[9*N+n]*b[4*N+n] + a[13*N+n]*b[7*N+n];
    c[ 14*N+n] = a[4*N+n]*b[2*N+n] + a[9*N+n]*b[5*N+n] + a[13*N+n]*b[8*N+n];
    c[ 15*N+n] = a[5*N+n]*b[0*N+n] + a[10*N+n]*b[3*N+n] + a[14*N+n]*b[6*N+n];
    c[ 16*N+n] = a[5*N+n]*b[1*N+n] + a[10*N+n]*b[4*N+n] + a[14*N+n]*b[7*N+n];
    c[ 17*N+n] = a[5*N+n]*b[2*N+n] + a[10*N+n]*b[5*N+n] + a[14*N+n]*b[8*N+n];
  }
}

template< typename TAcc>
__host__ __device__ inline void KalmanUpdate(MP6x6SF* trkErr, MP6F* inPar, const MP3x3SF* hitErr, const MP3F* msP, TAcc const & acc){
  MP3x3 inverse_temp;
  MP3x6 kGain;
  MP6x6SF newErr;
  KalmanGainInv(trkErr,hitErr,&inverse_temp,acc);
  KalmanGain(trkErr,&inverse_temp,&kGain,acc);

  using Dim = alpaka::dim::Dim<TAcc>;
  using Idx = alpaka::idx::Idx<TAcc>;
  using Vec = alpaka::vec::Vec<Dim, Idx>;

  Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
  Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
  for (size_t it=threadIdx[0];it<bsize;it+=threadExtent[0]) {
    const float xin = x(inPar,it);
    const float yin = y(inPar,it);
    const float zin = z(inPar,it);
    const float ptin = 1./ipt(inPar,it);
    const float phiin = phi(inPar,it);
    const float thetain = theta(inPar,it);
    const float xout = x(msP,it);
    const float yout = y(msP,it);
    const float zout = z(msP,it);

    float xnew = xin + (kGain.data[0*bsize+it]*(xout-xin)) +(kGain.data[1*bsize+it]*(yout-yin));
    float ynew = yin + (kGain.data[3*bsize+it]*(xout-xin)) +(kGain.data[4*bsize+it]*(yout-yin));
    float znew = zin + (kGain.data[6*bsize+it]*(xout-xin)) +(kGain.data[7*bsize+it]*(yout-yin));
    float ptnew = ptin + (kGain.data[9*bsize+it]*(xout-xin)) +(kGain.data[10*bsize+it]*(yout-yin));
    float phinew = phiin + (kGain.data[12*bsize+it]*(xout-xin)) +(kGain.data[13*bsize+it]*(yout-yin));
    float thetanew = thetain + (kGain.data[15*bsize+it]*(xout-xin)) +(kGain.data[16*bsize+it]*(yout-yin));

    newErr.data[0*bsize+it] = trkErr->data[0*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[0*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[2*bsize+it]);
    newErr.data[1*bsize+it] = trkErr->data[1*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[2*bsize+it] = trkErr->data[2*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[3*bsize+it] = trkErr->data[3*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[4*bsize+it] = trkErr->data[4*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[5*bsize+it] = trkErr->data[5*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[6*bsize+it] = trkErr->data[6*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[7*bsize+it] = trkErr->data[7*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[8*bsize+it] = trkErr->data[8*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[9*bsize+it] = trkErr->data[9*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[10*bsize+it] = trkErr->data[10*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[11*bsize+it] = trkErr->data[11*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[12*bsize+it] = trkErr->data[12*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[13*bsize+it] = trkErr->data[13*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[14*bsize+it] = trkErr->data[14*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[15*bsize+it] = trkErr->data[15*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[16*bsize+it] = trkErr->data[16*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[17*bsize+it] = trkErr->data[17*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[18*bsize+it] = trkErr->data[18*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[19*bsize+it] = trkErr->data[19*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[20*bsize+it] = trkErr->data[20*bsize+it] - (kGain.data[15*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[16*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[17*bsize+it]*trkErr->data[14*bsize+it]);

    setx(inPar,it,xnew );
    sety(inPar,it,ynew );
    setz(inPar,it,znew);
    setipt(inPar,it, ptnew);
    setphi(inPar,it, phinew);
    settheta(inPar,it, thetanew);
  }
  (*trkErr) = newErr;
 }

__device__ __constant__ float kfact= 100/3.8;
template< typename TAcc>
/*__host__*/ __device__ inline void propagateToZ(const MP6x6SF* inErr, const MP6F* inPar, const MP1I* inChg, const MP3F* msP,
	                MP6x6SF* outErr, MP6F* outPar, struct MP6x6F* errorProp, struct MP6x6F* temp, TAcc const & acc) {
  using Dim = alpaka::dim::Dim<TAcc>;
  using Idx = alpaka::idx::Idx<TAcc>;
  using Vec = alpaka::vec::Vec<Dim, Idx>;

  Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
  Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
  for (size_t it=threadIdx[0];it<bsize;it+=threadExtent[0]) {	
    const float zout = z(msP,it);
    const float k = q(inChg,it)*kfact;//100/3.8;
    const float deltaZ = zout - z(inPar,it);
    const float pt = 1./ipt(inPar,it);
    const float cosP = cosf(phi(inPar,it));
    const float sinP = sinf(phi(inPar,it));
    const float cosT = cosf(theta(inPar,it));
    const float sinT = sinf(theta(inPar,it));
    const float pxin = cosP*pt;
    const float pyin = sinP*pt;
    const float icosT = 1.0/cosT;
    const float icosTk = icosT/k;
    const float alpha = deltaZ*sinT*ipt(inPar,it)*icosTk;
    //const float alpha = deltaZ*sinT*ipt(inPar,it)/(cosT*k);
    const float sina = sinf(alpha); // this can be approximated;
    const float cosa = cosf(alpha); // this can be approximated;
    setx(outPar,it, x(inPar,it) + k*(pxin*sina - pyin*(1.-cosa)) );
    sety(outPar,it, y(inPar,it) + k*(pyin*sina + pxin*(1.-cosa)) );
    setz(outPar,it,zout);
    setipt(outPar,it, ipt(inPar,it));
    setphi(outPar,it, phi(inPar,it)+alpha );
    settheta(outPar,it, theta(inPar,it) );
    
    const float sCosPsina = sinf(cosP*sina);
    const float cCosPsina = cosf(cosP*sina);
    
    for (size_t i=0;i<6;++i) errorProp->data[bsize*PosInMtrx(i,i,6) + it] = 1.;
    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k*pt)*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ*(icosT*icosTk);
//    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)/cosT;
//    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*ipt(inPar,it))-k*(cosP*sina-sinP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
//    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k/ipt(inPar,it))*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
//    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*cosT);
//    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)/cosT;
//    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*ipt(inPar,it))-k*(sinP*sina+cosP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
//    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k/ipt(inPar,it))*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
//    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*cosT);
//    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT/(cosT*k);
//    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ/(cosT*k);
//    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ/(cosT*cosT*k);
  }
  //
  MultHelixPropEndcap(errorProp, inErr, temp,acc);
  MultHelixPropTranspEndcap(errorProp, temp, outErr,acc);
}





template< typename TAcc>
void ALPAKA_FN_ACC alpaka_kernel(TAcc const & acc, MPTRK* trk, MPHIT* hit, MPTRK* outtrk){
    using Dim = alpaka::dim::Dim<TAcc>;
    using Idx = alpaka::idx::Idx<TAcc>;
    using Vec = alpaka::vec::Vec<Dim, Idx>;

    Vec const threadIdx    = alpaka::idx::getIdx<alpaka::Block, alpaka::Threads>(acc);
    Vec const threadExtent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(acc);
    Vec const blockIdx    = alpaka::idx::getIdx<alpaka::Grid, alpaka::Blocks>(acc);
    Vec const blockExtent = alpaka::workdiv::getWorkDiv<alpaka::Grid, alpaka::Blocks>(acc);

   for (size_t ie=blockIdx[0];ie<nevts;ie+=blockExtent[0]) { // loop over events
     for (size_t ib=threadIdx[1];ib<nb;ib+=threadExtent[1]) { // loop over bunches of tracks
       const MPTRK* btracks = bTk(trk, ie, ib);
       MPTRK* obtracks = bTk(outtrk, ie, ib);
       for( size_t layer=0; layer<nlayer;++layer){
          const MPHIT* bhits = bHit(hit, ie, ib,layer);
 	        struct MP6x6F errorProp, temp;
          propagateToZ(&(*btracks).cov, &(*btracks).par, &(*btracks).q, &(*bhits).pos, &(*obtracks).cov, &(*obtracks).par,
	        &errorProp, &temp, acc); // vectorized function
          KalmanUpdate(&(*obtracks).cov,&(*obtracks).par,&(*bhits).cov,&(*bhits).pos,acc);
       }
     }
   }
}



void transfer(MPTRK* trk, MPHIT* hit, MPTRK* trk_dev, MPHIT* hit_dev){

  hipMemcpy(trk_dev, trk, nevts*nb*sizeof(MPTRK), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->par, &trk->par, sizeof(MP6F), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->par).data), &((trk->par).data), 6*bsize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->cov).data), &((trk->cov).data), 36*bsize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->q, &trk->q, sizeof(MP1I), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->q).data), &((trk->q).data), 1*bsize*sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(hit_dev,hit,nlayer*nevts*nb*sizeof(MPHIT), hipMemcpyHostToDevice);
  hipMemcpy(&hit_dev->pos,&hit->pos,sizeof(MP3F), hipMemcpyHostToDevice);
  hipMemcpy(&(hit_dev->pos).data,&(hit->pos).data,3*bsize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&hit_dev->cov,&hit->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice);
  hipMemcpy(&(hit_dev->cov).data,&(hit->cov).data,6*bsize*sizeof(float), hipMemcpyHostToDevice);
}

void transfer_back(MPTRK* trk, MPTRK* trk_host){
  hipMemcpy(trk_host, trk, nevts*nb*sizeof(MPTRK), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->par, &trk->par, sizeof(MP6F), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->par).data), &((trk->par).data), 6*bsize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->cov).data), &((trk->cov).data), 36*bsize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->q, &trk->q, sizeof(MP1I), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->q).data), &((trk->q).data), 1*bsize*sizeof(int), hipMemcpyDeviceToHost);
}
inline void transferTrk(MPTRK* trk, MPTRK* trk_dev){

  hipMemcpy(trk_dev, trk, nevts*nb*sizeof(MPTRK), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->par, &trk->par, sizeof(MP6F), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->par).data), &((trk->par).data), 6*bsize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->cov).data), &((trk->cov).data), 36*bsize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->q, &trk->q, sizeof(MP1I), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->q).data), &((trk->q).data), 1*bsize*sizeof(int), hipMemcpyHostToDevice);

}
inline void transferHit(MPHIT* hit, MPHIT* hit_dev){

    hipMemcpy(hit_dev,hit,nlayer*nevts*nb*sizeof(MPHIT), hipMemcpyHostToDevice);
    hipMemcpy(&hit_dev->pos,&hit->pos,sizeof(MP3F), hipMemcpyHostToDevice);
    hipMemcpy(&(hit_dev->pos).data,&(hit->pos).data,3*bsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&hit_dev->cov,&hit->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice);
    hipMemcpy(&(hit_dev->cov).data,&(hit->cov).data,6*bsize*sizeof(float), hipMemcpyHostToDevice);
}







int main (int argc, char* argv[]) {

  using Dim = alpaka::dim::DimInt<2u>;
  using Idx = std::size_t;
  // set type of accelerator
  using Acc = alpaka::acc::AccGpuCudaRt<Dim, Idx>;

  using DevAcc = alpaka::dev::Dev<Acc>;
  using PltfAcc = alpaka::pltf::Pltf<DevAcc>;

  using QueueProperty = alpaka::queue::Blocking;
  using QueueAcc = alpaka::queue::Queue<Acc,QueueProperty>;


  using Host = alpaka::acc::AccCpuSerial<Dim, Idx>;
  using HostQueueProperty = alpaka::queue::Blocking;
  using QueueHost = alpaka::queue::Queue<Host, HostQueueProperty>;
  using DevHost = alpaka::dev::Dev<Host>;
  using PltfHost = alpaka::pltf::Pltf<DevHost>;
  
  // select device
  DevAcc const devAcc(alpaka::pltf::getDevByIdx<PltfAcc>(0u));
  DevHost const devHost(alpaka::pltf::getDevByIdx<PltfHost>(0u));

  //make queue on device
  QueueAcc accQueue(devAcc);
  QueueHost hostQueue(devHost);

  using Vec = alpaka::vec::Vec<Dim,Idx>;
  //Vec const elementsPerThread(Vec::all(static_cast<Idx>(4)));
  //Vec const threadsPerBlock(Vec::all(static_cast<Idx>(8)));
  //Vec const blocksPerGrid(static_cast<Idx>(4),static_cast<Idx>(1));//,static_cast<Idx>(2));
  //static constexpr uint64_t blockSize = alpaka::dim::DimInt<2>::value; 
  //Idx blockCount = static_cast<Idx>(alpaka::acc::getAccDevProps<Acc,DevAcc>(devAcc).m_multiProcessorCount*8);

  Vec const elementsPerThread(Vec::all(static_cast<Idx>(32)));
  //Vec const threadsPerBlock(Vec::all(static_cast<Idx>(8)));
  Vec const threadsPerBlock(static_cast<Idx>(16),static_cast<Idx>(16));//,static_cast<Idx>(1));
  //Vec const threadsPerBlock(Vec::all(static_cast<Idx>(1)));
  Vec const blocksPerGrid(static_cast<Idx>(20),static_cast<Idx>(1));//,static_cast<Idx>(1));
  //Vec const blocksPerGrid(Vec::all(static_cast<Idx>(1)));
  //Vec const blocksPerGrid(static_cast<Idx>(4),static_cast<Idx>(4),static_cast<Idx>(4));
  //Vec const extent = alpaka::workdiv::getWorkDiv<alpaka::Block, alpaka::Threads>(Host);

  using WorkDiv = alpaka::workdiv::WorkDivMembers<Dim, Idx>;
  //WorkDiv const workDiv( static_cast<Idx>(blockCount), static_cast<Idx>(blockSize),block);
  //WorkDiv workDiv{ static_cast<Idx>(blockCount), static_cast<Idx>(blockSize),static_cast<Idx>(1)};
  //WorkDiv const workDiv( blocksPerGrid, static_cast<Idx>(blockSize),elementsPerThread);
  WorkDiv const workDiv( blocksPerGrid, threadsPerBlock,elementsPerThread);



   int itr;
   ATRK inputtrk = {
     {-12.806846618652344, -7.723824977874756, 38.13014221191406,0.23732035065189902, -2.613372802734375, 0.35594117641448975},
     {6.290299552347278e-07,4.1375109560704004e-08,7.526661534029699e-07,2.0973730840978533e-07,1.5431574240665213e-07,9.626245400795597e-08,-2.804026640189443e-06,
      6.219111130687595e-06,2.649119409845118e-07,0.00253512163402557,-2.419662877381737e-07,4.3124190760040646e-07,3.1068903991780678e-09,0.000923913115050627,
      0.00040678296006807003,-7.755406890332818e-07,1.68539375883925e-06,6.676875566525437e-08,0.0008420574605423793,7.356584799406111e-05,0.0002306247719158348},
     1
   };

   AHIT inputhit = {
     {-20.7824649810791, -12.24150276184082, 57.8067626953125},
     {2.545517190810642e-06,-2.6680759219743777e-06,2.8030024168401724e-06,0.00014160551654640585,0.00012282167153898627,11.385087966918945}
   };

   printf("track in pos: %f, %f, %f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2]);
   printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66(PosInMtrx(0,0,6))],
	                                       inputtrk.cov[SymOffsets66(PosInMtrx(1,1,6))],
	                                       inputtrk.cov[SymOffsets66(PosInMtrx(2,2,6))]);
   printf("hit in pos: %f %f %f \n", inputhit.pos[0], inputhit.pos[1], inputhit.pos[2]);
   
   printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
   printf("NITER=%d\n", NITER);

   long setup_start, setup_stop;
   struct timeval timecheck;

   gettimeofday(&timecheck, NULL);
   setup_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
#ifdef FIXED_RSEED
   //[DEBUG by Seyong on Dec. 28, 2020] add an explicit srand(1) call to generate fixed inputs for better debugging.
   srand(1);
#endif
   MPTRK* trk = prepareTracks(inputtrk);
   MPHIT* hit = prepareHits(inputhit);
   MPTRK* outtrk = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK));
   MPTRK* trk_dev;
   MPHIT* hit_dev;
   hipMalloc((MPTRK**)&trk_dev,nevts*nb*sizeof(MPTRK));
   hipMalloc((MPHIT**)&hit_dev,nlayer*nevts*nb*sizeof(MPHIT));
   MPTRK* outtrk_dev;
   hipMalloc((MPTRK**)&outtrk_dev,nevts*nb*sizeof(MPTRK));
   //MPTRK* outtrkx = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK));

   
//old memory transfer attemp

//    constexpr Idx nElementsPerDim = 4;///*nevts*nb*/sizeof(MPTRK);
//    //const Vec extents(Vec::all(static_cast<Idx>(nElementsPerDim)));
//    const Vec extents(Vec::all(static_cast<std::uint32_t>(nElementsPerDim)));
//   using ViewHosttrk = alpaka::mem::view::ViewPlainPtr<DevHost, MPTRK, Dim, Idx>;
//   ViewHosttrk hostViewPlainPtr_trk(trk, devHost,extents);
//   MPTRK* const trk_host = alpaka::mem::view::getPtrNative(hostViewPlainPtr_trk);
//   ViewHosttrk hostViewPlainPtr_outtrk(outtrkx, devHost,extents);
//   MPTRK* const outtrk_host = alpaka::mem::view::getPtrNative(hostViewPlainPtr_outtrk);
//   ViewHosttrk hostViewPlainPtr_outtrk_fin(outtrk, devHost,extents);
//   MPTRK* const outtrk_hostfin = alpaka::mem::view::getPtrNative(hostViewPlainPtr_outtrk_fin);
//   using ViewHosthit = alpaka::mem::view::ViewPlainPtr<DevHost, MPHIT, Dim, Idx>;
//   ViewHosthit hostViewPlainPtr_hit(hit, devHost,extents);
//   MPHIT* const hit_host = alpaka::mem::view::getPtrNative(hostViewPlainPtr_hit);
//
//
//
//
//   using BufAcctrk = alpaka::mem::buf::Buf<DevAcc,MPTRK,Dim,Idx>;
//   using BufAcchit = alpaka::mem::buf::Buf<DevAcc,MPHIT,Dim,Idx>;
//   BufAcctrk trkbuf(alpaka::mem::buf::alloc<MPTRK,Idx>(devAcc,extents));
//   BufAcctrk outtrkbuf(alpaka::mem::buf::alloc<MPTRK,Idx>(devAcc,extents));
//   //BufAcctrk trkbuf(alpaka::mem::buf::alloc<MPTRK,Idx>(devAcc,nevts*nb*sizeof(MPTRK)));
//   //BufAcchit hitbuf(alpaka::mem::buf::alloc<MPHIT,Idx>(devAcc,nevts*nb*sizeof(MPHIT)));
//   BufAcchit hitbuf(alpaka::mem::buf::alloc<MPHIT,Idx>(devAcc,extents));
//
//   MPTRK* outtrk_devx;
//   hipMalloc((MPTRK**)&outtrk_devx,nevts*nb*sizeof(MPTRK));
//   using ViewDevouttrk = alpaka::mem::view::ViewPlainPtr<DevAcc, MPTRK, Dim, Idx>;
//   ViewDevouttrk hostViewPlainPtr_outtrkdev(outtrk_devx, devAcc,extents);
//   //MPTRK* const trk_host = alpaka::mem::view::getPtrNative(hostViewPlainPtr_trk);
//   //MPHIT* hit_dev;
//   //hipMalloc((MPHIT**)&hit_dev,nevts*nb*sizeof(MPHIT));
//   //using ViewDevtrk = alpaka::mem::view::ViewPlainPtr<DevAcc, MPHIT, Dim, Idx>;
//   //ViewDevtrk hitbuf(hit_dev, devAcc,extents);
//   //MPTRK* const trkbuf = alpaka::mem::view::getPtrNative(trkbufx);
//   //MPTRK* trkbuf = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK)); //fixme, align?
//   alpaka::mem::view::copy(accQueue, hitbuf, hostViewPlainPtr_hit,extents);
//   alpaka::mem::view::copy(accQueue, trkbuf, hostViewPlainPtr_trk,extents);
//   //alpaka::mem::view::copy(devQueue, trkbuf, hostViewPlainPtr_trk,nevts*nb*sizeof(MPTRK));
//   //alpaka::mem::view::copy(devQueue, hitbuf, hostViewPlainPtr_hit,nevts*nb*sizeof(MPHIT));
//
//   MPTRK * const trk_dev = alpaka::mem::view::getPtrNative(trkbuf);
//   MPTRK * const outtrk_dev = alpaka::mem::view::getPtrNative(hostViewPlainPtr_outtrkdev);
//   //MPTRK * const outtrk_dev = alpaka::mem::view::getPtrNative(outtrkbuf);
//   MPHIT * const hit_dev = alpaka::mem::view::getPtrNative(hitbuf);

   gettimeofday(&timecheck, NULL);
   setup_stop = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

   printf("done preparing!\n");
   
//old memory transfer attemp  

  //using Data_hit = MPHIT;
  //using Data_trk = MPTRK;
  //using Dim = alpaka::dim::DimInt<1>;
  //using Idx = std::size_t;
  //using BufHost_hit = alpaka::mem::buf::Buf<DevAcc,Data_hit,Dim,Idx>;
  //using BufHost_trk = alpaka::mem::buf::Buf<DevAcc,Data_trk,Dim,Idx>;
  //BufHost_hit bufhit_dev(alpaka::mem::buf::alloc<Data_hit, Idx>(devAcc, nevts*nb*sizeof(MPHIT)));
  //BufHost_trk buftrk_dev(alpaka::mem::buf::alloc<Data_trk, Idx>(devAcc, nevts*nb*sizeof(MPTRK)));
  //BufHost_trk bufouttrk_dev(alpaka::mem::buf::alloc<Data_trk, Idx>(devAcc, nevts*nb*sizeof(MPTRK)));
  //using DevHost = alpaka::dev::DevCpu;
  //using PltfHost = alpaka::pltf::Pltf<DevHost>;
  //DevHost const devHost(alpaka::pltf::getDevByIdx<PltfHost>(0u));
  //BufHost_trk bufouttrk(alpaka::mem::buf::alloc<Data_trk, Idx>(devHost, nevts*nb*sizeof(MPTRK)));
  //Data_trk * outtrk_dev(alpaka::mem::view::getPtrNative(bufouttrk_dev));
  //Data_trk * outtrk(alpaka::mem::view::getPtrNative(bufouttrk));
  //Data_trk * trk_dev(alpaka::mem::view::getPtrNative(buftrk_dev));
  //Data_hit * hit_dev(alpaka::mem::view::getPtrNative(bufhit_dev));


   printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
   printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
   printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*nb*sizeof(struct MPHIT));

  //copy host to acc
  //alpaka::mem::view::copy(queue,trk_dev,trk,nevts*nb*sizeof(MPTRK));
  //alpaka::mem::view::copy(queue,trk_dev->par,trk->par,sizeof(MP6F));
  //alpaka::mem::view::copy(queue,hit_dev,hit,nevts*nb*sizeof(MPHIT));

   //transfer(trk,hit, trk_dev,hit_dev);

   auto wall_start = std::chrono::high_resolution_clock::now();

   for(itr=0; itr<NITER; itr++) {
   transferTrk(trk,trk_dev);
   //for(int layer=0; layer<nlayer; layer++) {
   transferHit(hit,hit_dev);
   //transfer(trk,hit, trk_dev,hit_dev);
     //alpaka::kernel::exec<Host>( hostQueue,workDiv,
     //[] ALPAKA_FN_ACC (Host const & host, MPTRK* trk_host, MPHIT* hit_host, MPTRK* outtrk_host){
     //alpaka_kernel(host, trk_host,hit_host,outtrk_host);
     //}, trk_host, hit_host, outtrk_host);
     //alpaka::wait::wait(hostQueue);
     

    alpaka::kernel::exec<Acc>( accQueue,workDiv,
     [] ALPAKA_FN_ACC (Acc const & acc, MPTRK* trk_dev, MPHIT* hit_dev, MPTRK* outtrk_dev){
     alpaka_kernel(acc, trk_dev,hit_dev,outtrk_dev);
     }, trk_dev, hit_dev, outtrk_dev);

     alpaka::wait::wait(accQueue);
     transfer_back(outtrk_dev,outtrk);
  } //end of itr loop
  
   hipDeviceSynchronize(); 
   auto wall_stop = std::chrono::high_resolution_clock::now();
  
//}
   //alpaka::mem::view::copy(accQueue, hostViewPlainPtr_outtrk_fin,hostViewPlainPtr_outtrkdev,extents);
   //alpaka::wait::wait(accQueue);
   //alpaka::mem::view::copy(hostQueue, hostViewPlainPtr_outtrk_fin,hostViewPlainPtr_outtrk,extents);
   //alpaka::wait::wait(hostQueue);
  // transfer_back(outtrk_dev,outtrk);

   auto wall_diff = wall_stop - wall_start;
   auto wall_time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
   printf("setup time time=%f (s)\n", (setup_stop-setup_start)*0.001);
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), wall_time, wall_time/(nevts*ntrks*int(NITER)));
   printf("formatted %i %i %i %i %i %f 0 %f %i\n",int(NITER),nevts, ntrks, bsize, nb, wall_time, (setup_stop-setup_start)*0.001, 3);


   float avgx = 0, avgy = 0, avgz = 0;
   float avgpt = 0, avgphi = 0, avgtheta = 0;
   float avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
     }
   }
   avgpt = avgpt/float(nevts*ntrks);
   avgphi = avgphi/float(nevts*ntrks);
   avgtheta = avgtheta/float(nevts*ntrks);
   avgx = avgx/float(nevts*ntrks);
   avgy = avgy/float(nevts*ntrks);
   avgz = avgz/float(nevts*ntrks);
   avgdx = avgdx/float(nevts*ntrks);
   avgdy = avgdy/float(nevts*ntrks);
   avgdz = avgdz/float(nevts*ntrks);

   float stdx = 0, stdy = 0, stdz = 0;
   float stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
     }
   }

   stdx = sqrtf(stdx/float(nevts*ntrks));
   stdy = sqrtf(stdy/float(nevts*ntrks));
   stdz = sqrtf(stdz/float(nevts*ntrks));
   stddx = sqrtf(stddx/float(nevts*ntrks));
   stddy = sqrtf(stddy/float(nevts*ntrks));
   stddz = sqrtf(stddz/float(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);

//   free(trk);
//   free(hit);
//   free(outtrk);

   return 0;
}
